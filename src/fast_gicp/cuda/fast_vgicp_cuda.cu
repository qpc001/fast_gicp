#include "hip/hip_runtime.h"
#include <fast_gicp/cuda/fast_vgicp_cuda.cuh>

#include <thrust/device_new.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <fast_gicp/cuda/brute_force_knn.cuh>
#include <fast_gicp/cuda/covariance_estimation.cuh>
#include <fast_gicp/cuda/covariance_regularization.cuh>
#include <fast_gicp/cuda/gaussian_voxelmap.cuh>
#include <fast_gicp/cuda/compute_mahalanobis.cuh>
#include <fast_gicp/cuda/compute_derivatives.cuh>
#include <fast_gicp/cuda/find_voxel_correspondences.cuh>

namespace fast_gicp {
namespace cuda {

FastVGICPCudaCore::FastVGICPCudaCore() {
  // warming up GPU
  hipDeviceSynchronize();

  resolution = 1.0;
  linearized_x.setIdentity();

  kernel_width = 0.25;
  kernel_max_dist = 3.0;

  offsets.reset(new thrust::device_vector<Eigen::Vector3i>(1));
  (*offsets)[0] = Eigen::Vector3i::Zero().eval();
}
FastVGICPCudaCore ::~FastVGICPCudaCore() {}

void FastVGICPCudaCore::set_resolution(double resolution) {
  this->resolution = resolution;
}

void FastVGICPCudaCore::set_kernel_params(double kernel_width, double kernel_max_dist) {
  this->kernel_width = kernel_width;
  this->kernel_max_dist = kernel_max_dist;
}

void FastVGICPCudaCore::set_neighbor_search_method(fast_gicp::NeighborSearchMethod method, double radius) {
  thrust::host_vector<Eigen::Vector3i, Eigen::aligned_allocator<Eigen::Vector3i>> h_offsets;

  switch (method) {
    default:
      std::cerr << "here must not be reached" << std::endl;
      abort();

    case fast_gicp::NeighborSearchMethod::DIRECT1:
      h_offsets.resize(1);
      h_offsets[0] = Eigen::Vector3i::Zero();
      break;

    case fast_gicp::NeighborSearchMethod::DIRECT7:
      h_offsets.resize(7);
      h_offsets[0] = Eigen::Vector3i(0, 0, 0);
      h_offsets[1] = Eigen::Vector3i(1, 0, 0);
      h_offsets[2] = Eigen::Vector3i(-1, 0, 0);
      h_offsets[3] = Eigen::Vector3i(0, 1, 0);
      h_offsets[4] = Eigen::Vector3i(0, -1, 0);
      h_offsets[5] = Eigen::Vector3i(0, 0, 1);
      h_offsets[6] = Eigen::Vector3i(0, 0, -1);
      break;

    case fast_gicp::NeighborSearchMethod::DIRECT27:
      h_offsets.reserve(27);
      for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
          for (int k = 0; k < 3; k++) {
            h_offsets.push_back(Eigen::Vector3i(i - 1, j - 1, k - 1));
          }
        }
      }
      break;

    case fast_gicp::NeighborSearchMethod::DIRECT_RADIUS:
      h_offsets.reserve(50);
      int range = std::ceil(radius);
      for (int i = -range; i <= range; i++) {
        for (int j = -range; j <= range; j++) {
          for (int k = -range; k <= range; k++) {
            Eigen::Vector3i offset(i, j, k);
            if(offset.cast<double>().norm() <= radius + 1e-3) {
              h_offsets.push_back(offset);
            }
          }
        }
      }

      break;
  }

  *offsets = h_offsets;
}

void FastVGICPCudaCore::swap_source_and_target() {
  source_points.swap(target_points);
  source_neighbors.swap(target_neighbors);
  source_covariances.swap(target_covariances);

  if(!target_points || !target_covariances) {
    return;
  }

  create_target_voxelmap();
}

/**
 * @brief 传入std::vector形式的点云，将数据保存到显存上的数据thrust::device_vector
 * @param cloud
 */
void FastVGICPCudaCore::set_source_cloud(const std::vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>>& cloud) {
  // 先构造thrust::host_vector，并以std::vector形式的点云作为数据输入
  thrust::host_vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>> points(cloud.begin(), cloud.end());
  // 检查source_points是否为空
  if(!source_points) {
    source_points.reset(new Points());
  }

  // 将thrust::host_vector转成显存上的数据thrust::device_vector
  *source_points = points;
}

/**
 * @brief 传入std::vector形式的点云，将数据保存到显存上的数据thrust::device_vector
 * @param cloud
 */
void FastVGICPCudaCore::set_target_cloud(const std::vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>>& cloud) {
  // 先构造thrust::host_vector，并以std::vector形式的点云作为数据输入
  thrust::host_vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>> points(cloud.begin(), cloud.end());
  if(!target_points) {
    target_points.reset(new Points());
  }

  // 将thrust::host_vector转成显存上的数据thrust::device_vector
  *target_points = points;
}

/**
 * @brief 设置源点云的近邻点的索引，并且保存到显存设备上thrust::device_vector
 * @param k
 * @param neighbors
 */
void FastVGICPCudaCore::set_source_neighbors(int k, const std::vector<int>& neighbors) {
  assert(k * source_points->size() == neighbors.size());
  // 首先在thrust::host_vector分配
  thrust::host_vector<int> k_neighbors(neighbors.begin(), neighbors.end());

  if(!source_neighbors) {
    source_neighbors.reset(new thrust::device_vector<int>());
  }

  // 然后保存到thrust::device_vector显存设备上
  *source_neighbors = k_neighbors;
}

void FastVGICPCudaCore::set_target_neighbors(int k, const std::vector<int>& neighbors) {
  assert(k * target_points->size() == neighbors.size());
  thrust::host_vector<int> k_neighbors(neighbors.begin(), neighbors.end());

  if(!target_neighbors) {
    target_neighbors.reset(new thrust::device_vector<int>());
  }

  *target_neighbors = k_neighbors;
}

struct untie_pair_second {
  __device__ int operator()(thrust::pair<float, int>& p) const {
    return p.second;
  }
};

/**
 * @brief 对于源点云中的每个点x，遍历目标点云，找到关于源点云中的一个点x的目标点云近邻点索引，结果保存到成员变量
 * @param k
 */
void FastVGICPCudaCore::find_source_neighbors(int k) {
  assert(source_points);

  // 对于源点云中的每个点x，遍历目标点云，找到关于源点云中的一个点x的k个近邻点索引(在目标点云中的)，结果保存到k_neighbors
  thrust::device_vector<thrust::pair<float, int>> k_neighbors;
  brute_force_knn_search(*source_points, *source_points, k, k_neighbors);

  if(!source_neighbors) {
    source_neighbors.reset(new thrust::device_vector<int>(k_neighbors.size()));
  } else {
    source_neighbors->resize(k_neighbors.size());
  }

  // 对于k_neighbors中的每个元素，提取其second元素，保存到source_neighbors容器（显存上）
  thrust::transform(k_neighbors.begin(), k_neighbors.end(), source_neighbors->begin(), untie_pair_second());
}

void FastVGICPCudaCore::find_target_neighbors(int k) {
  assert(target_points);

  thrust::device_vector<thrust::pair<float, int>> k_neighbors;
  brute_force_knn_search(*target_points, *target_points, k, k_neighbors);

  if(!target_neighbors) {
    target_neighbors.reset(new thrust::device_vector<int>(k_neighbors.size()));
  } else {
    target_neighbors->resize(k_neighbors.size());
  }
  thrust::transform(k_neighbors.begin(), k_neighbors.end(), target_neighbors->begin(), untie_pair_second());
}

/**
 * @brief 计算并重组协方差矩阵
 * @param method
 */
void FastVGICPCudaCore::calculate_source_covariances(RegularizationMethod method) {
  assert(source_points && source_neighbors);
  // 计算k值，即每个点有多少个对应的近邻点
  int k = source_neighbors->size() / source_points->size();

  if(!source_covariances) {
    source_covariances.reset(new thrust::device_vector<Eigen::Matrix3f>(source_points->size()));
  }

  // 计算协方差，source_covariances的结果与source_points索引一一对应
  covariance_estimation(*source_points, k, *source_neighbors, *source_covariances);
  // 对协方差进行regularization（默认使用平面性质，第三个特征值很小的特性）
  covariance_regularization(*source_points, *source_covariances, method);
}

void FastVGICPCudaCore::calculate_target_covariances(RegularizationMethod method) {
  assert(target_points && target_neighbors);
  int k = target_neighbors->size() / target_points->size();

  if(!target_covariances) {
    target_covariances.reset(new thrust::device_vector<Eigen::Matrix3f>(target_points->size()));
  }
  covariance_estimation(*target_points, k, *target_neighbors, *target_covariances);
  covariance_regularization(*target_points, *target_covariances, method);
}

void FastVGICPCudaCore::calculate_source_covariances_rbf(RegularizationMethod method) {
  if(!source_covariances) {
    source_covariances.reset(new thrust::device_vector<Eigen::Matrix3f>(source_points->size()));
  }
  covariance_estimation_rbf(*source_points, kernel_width, kernel_max_dist, *source_covariances);
  covariance_regularization(*source_points, *source_covariances, method);
}

void FastVGICPCudaCore::calculate_target_covariances_rbf(RegularizationMethod method) {
  if(!target_covariances) {
    target_covariances.reset(new thrust::device_vector<Eigen::Matrix3f>(target_points->size()));
  }
  covariance_estimation_rbf(*target_points, kernel_width, kernel_max_dist, *target_covariances);
  covariance_regularization(*target_points, *target_covariances, method);
}

void FastVGICPCudaCore::get_voxel_correspondences(std::vector<std::pair<int, int>>& correspondences) const {
  thrust::host_vector<thrust::pair<int, int>> corrs = *voxel_correspondences;
  correspondences.resize(corrs.size());
  std::transform(corrs.begin(), corrs.end(), correspondences.begin(), [](const auto& x) { return std::make_pair(x.first, x.second); });
}

void FastVGICPCudaCore::get_voxel_num_points(std::vector<int>& num_points) const {
  thrust::host_vector<int> voxel_num_points = voxelmap->num_points;
  num_points.resize(voxel_num_points.size());
  std::copy(voxel_num_points.begin(), voxel_num_points.end(), num_points.begin());
}

void FastVGICPCudaCore::get_voxel_means(std::vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>>& means) const {
  thrust::host_vector<Eigen::Vector3f, Eigen::aligned_allocator<Eigen::Vector3f>> voxel_means = voxelmap->voxel_means;
  means.resize(voxel_means.size());
  std::copy(voxel_means.begin(), voxel_means.end(), means.begin());
}

void FastVGICPCudaCore::get_voxel_covs(std::vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>>& covs) const {
  thrust::host_vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>> voxel_covs = voxelmap->voxel_covs;
  covs.resize(voxel_covs.size());
  std::copy(voxel_covs.begin(), voxel_covs.end(), covs.begin());
}

void FastVGICPCudaCore::get_source_covariances(std::vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>>& covs) const {
  thrust::host_vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>> c = *source_covariances;
  covs.resize(c.size());
  std::copy(c.begin(), c.end(), covs.begin());
}

void FastVGICPCudaCore::get_target_covariances(std::vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>>& covs) const {
  thrust::host_vector<Eigen::Matrix3f, Eigen::aligned_allocator<Eigen::Matrix3f>> c = *target_covariances;
  covs.resize(c.size());
  std::copy(c.begin(), c.end(), covs.begin());
}

void FastVGICPCudaCore::create_target_voxelmap() {
  assert(target_points && target_covariances);
  if(!voxelmap) {
    voxelmap.reset(new GaussianVoxelMap(resolution));
  }
  voxelmap->create_voxelmap(*target_points, *target_covariances);
}

void FastVGICPCudaCore::update_correspondences(const Eigen::Isometry3d& trans) {
  thrust::device_vector<Eigen::Isometry3f> trans_ptr(1);
  trans_ptr[0] = trans.cast<float>();

  if(voxel_correspondences == nullptr) {
    voxel_correspondences.reset(new Correspondences());
  }
  linearized_x = trans.cast<float>();
  find_voxel_correspondences(*source_points, *voxelmap, trans_ptr.data(), *offsets, *voxel_correspondences);
}

double FastVGICPCudaCore::compute_error(const Eigen::Isometry3d& trans, Eigen::Matrix<double, 6, 6>* H, Eigen::Matrix<double, 6, 1>* b) const {
  thrust::host_vector<Eigen::Isometry3f, Eigen::aligned_allocator<Eigen::Isometry3f>> trans_(2);
  trans_[0] = linearized_x;
  trans_[1] = trans.cast<float>();

  thrust::device_vector<Eigen::Isometry3f> trans_ptr = trans_;

  return compute_derivatives(*source_points, *source_covariances, *voxelmap, *voxel_correspondences, trans_ptr.data(), trans_ptr.data() + 1, H, b);
}

}  // namespace cuda
}  // namespace fast_gicp
